#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Kernel definition
__global__ void what_is_my_id(unsigned int * const block,
                              unsigned int * const thread,
                              unsigned int * const warp,
                              unsigned int * const calc_thread) {
    // Thread id is block index * block size + thread offset into the block
    const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    block[thread_idx] = blockIdx.x;
    thread[thread_idx] = threadIdx.x;

    // Calculate warp using the built-in variable warpSize
    warp[thread_idx] = threadIdx.x / warpSize;
    calc_thread[thread_idx] = thread_idx;
}

#define ARRAY_SIZE 128
#define ARRAY_SIZE_IN_BYTES (sizeof(unsigned int) * ARRAY_SIZE)

// Declare statically four arrays of ARRAY_SIZE each
unsigned int cpu_block[ARRAY_SIZE];
unsigned int cpu_thread[ARRAY_SIZE];
unsigned int cpu_warp[ARRAY_SIZE];
unsigned int cpu_calc_thread[ARRAY_SIZE];

int main(void) {
    // Total thread count = 2 * 64 = 128
    const unsigned int num_blocks = 2;
    const unsigned int num_threads = 64;

    // Declare pointers for GPU-based params
    unsigned int *gpu_block;
    unsigned int *gpu_thread;
    unsigned int *gpu_warp;
    unsigned int *gpu_calc_thread;

    // Allocate four arrays on the GPU
    hipMalloc((void **)&gpu_block, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_warp, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);

    // Execute our kernel
    what_is_my_id<<<num_blocks, num_threads>>>(gpu_block, gpu_thread, gpu_warp, gpu_calc_thread);

    // Copy back the GPU results to the CPU
    hipMemcpy(cpu_block, gpu_block, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_warp, gpu_warp, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
    hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

    // Free the arrays on the GPU as now we’re done with them
    hipFree(gpu_block);
    hipFree(gpu_thread);
    hipFree(gpu_warp);
    hipFree(gpu_calc_thread);

    // Iterate through the arrays and print
    for (unsigned int i = 0; i < ARRAY_SIZE; i++) {
        printf("Calculated Thread: %3u - Block: %2u - Warp: %2u - Thread: %3u\n",
               cpu_calc_thread[i], cpu_block[i], cpu_warp[i], cpu_thread[i]);
    }

    return 0;
}
