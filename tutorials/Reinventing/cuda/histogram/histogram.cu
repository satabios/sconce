#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "cuda_common.cuh"

#define NUM_BINS 256

__global__ void histogram_kernel(const unsigned char* input, unsigned int* histogram, int size) { 
  __shared__ unsigned int local_histogram[NUM_BINS];

  for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
    local_histogram[i] = 0;
  }
  __syncthreads();

  int global_start = blockIdx.x * blockDim.x  + threadIdx.x;
  int stride = blockDim.x * gridDim.x ;

  for (int i = global_start; i < size; i += stride) {
      unsigned char value = input[i ];
      atomicAdd(&local_histogram[value], 1);
  
  }
  __syncthreads();

  for (int i = threadIdx.x; i < NUM_BINS; i += blockDim.x) {
    atomicAdd(&histogram[i], local_histogram[i]);
  }
}



int main() {
  int width = 512;
  int height = 512;
  int size = width * height;

  unsigned char* image = (unsigned char*)malloc(size * sizeof(unsigned char));
  unsigned int* bins = (unsigned int*)malloc(NUM_BINS * sizeof(unsigned int));

  for (int i = 0; i < size; i++) {
    image[i] = rand() % 256;
  }

  unsigned char* d_image;
  unsigned int* d_bins;
  CUDA_CHECK(hipMalloc((void**)&d_image, size * sizeof(unsigned char)));
  CUDA_CHECK(hipMalloc((void**)&d_bins, NUM_BINS * sizeof(unsigned int)));

  CUDA_CHECK(hipMemcpy(d_image, image, size * sizeof(unsigned char), hipMemcpyHostToDevice));

  CUDA_CHECK(hipMemset(d_bins, 0, NUM_BINS * sizeof(unsigned int))); 
  
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));

  CUDA_CHECK(hipEventRecord(start));

  int threadsPerBlock = 1024; // Higher thread count for coarsening
  int blocksPerGrid = (size + threadsPerBlock - 1) / (threadsPerBlock);
  histogram_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_image, d_bins, size);
  CUDA_CHECK(hipDeviceSynchronize());

  CUDA_CHECK(hipEventRecord(stop));
  CUDA_CHECK(hipEventSynchronize(stop));

  float milliseconds = 0;
  CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));

  printf("Kernel execution time: %.3f ms\n", milliseconds);

  CUDA_CHECK(hipMemcpy(bins, d_bins, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost));

  free(image);
  free(bins);
  CUDA_CHECK(hipFree(d_image));
  CUDA_CHECK(hipFree(d_bins));

  return 0;
}
