#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
__global__ void what_is_my_id_2d_A(
    unsigned int * const block_x,
    unsigned int * const block_y,
    unsigned int * const thread,
    unsigned int * const calc_thread,
    unsigned int * const x_thread,
    unsigned int * const y_thread,
    unsigned int * const grid_dimx,
    unsigned int * const block_dimx,
    unsigned int * const grid_dimy,
    unsigned int * const block_dimy)
{
    const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const unsigned int thread_idx = (gridDim.x * blockDim.x) * idy + idx;
    block_x[thread_idx] = blockIdx.x;
    block_y[thread_idx] = blockIdx.y;
    thread[thread_idx] = threadIdx.x;
    calc_thread[thread_idx] = thread_idx;
    x_thread[thread_idx] = idx;
    y_thread[thread_idx] = idy;
    grid_dimx[thread_idx] = gridDim.x;
    block_dimx[thread_idx] = blockDim.x;
    grid_dimy[thread_idx] = gridDim.y;
    block_dimy[thread_idx] = blockDim.y;
}

#define ARRAY_SIZE_X 32
#define ARRAY_SIZE_Y 16
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * sizeof(unsigned int))

unsigned int cpu_block_x[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_y[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_warp[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_calc_thread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_xthread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_ythread[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimx[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_grid_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];
unsigned int cpu_block_dimy[ARRAY_SIZE_Y][ARRAY_SIZE_X];

int main(void){
    const dim3 threads_rect(32, 4);
    const dim3 blocks_rect(1, 4);

    const dim3 threads_square(16, 8);
    const dim3 blocks_square(2, 2);

    char ch;

    unsigned int * gpu_block_x;
    unsigned int * gpu_block_y;
    unsigned int * gpu_thread;
    unsigned int * gpu_warp;
    unsigned int * gpu_calc_thread;
    unsigned int * gpu_xthread;
    unsigned int * gpu_ythread;
    unsigned int * gpu_grid_dimx;
    unsigned int * gpu_block_dimx;
    unsigned int * gpu_grid_dimy;
    unsigned int * gpu_block_dimy;

    hipMalloc((void **)&gpu_block_x, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_y, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_calc_thread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_xthread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_ythread, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimx, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_grid_dimy, ARRAY_SIZE_IN_BYTES);
    hipMalloc((void **)&gpu_block_dimy, ARRAY_SIZE_IN_BYTES);

    for (int kernel = 0; kernel < 2; kernel++){
        switch (kernel)
        {
        case 0:
        {
            what_is_my_id_2d_A << <blocks_rect, threads_rect >> >(gpu_block_x, gpu_block_y,
                gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx,
                gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
        } break;

        case 1:
        {
            what_is_my_id_2d_A << <blocks_square, threads_square >> >(gpu_block_x, gpu_block_y,
                gpu_thread, gpu_calc_thread, gpu_xthread, gpu_ythread, gpu_grid_dimx,
                gpu_block_dimx, gpu_grid_dimy, gpu_block_dimy);
        } break;

        default: exit(1); break;
        }

        hipMemcpy(cpu_block_x, gpu_block_x, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_y, gpu_block_y, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_thread, gpu_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_calc_thread, gpu_calc_thread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_xthread, gpu_xthread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_ythread, gpu_ythread, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimx, gpu_grid_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimx, gpu_block_dimx, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_grid_dimy, gpu_grid_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);
        hipMemcpy(cpu_block_dimy, gpu_block_dimy, ARRAY_SIZE_IN_BYTES, hipMemcpyDeviceToHost);

        printf("\nKernel %d\n", kernel);
        for (int y = 0; y < ARRAY_SIZE_Y; y++){
            for (int x = 0; x < ARRAY_SIZE_X; x++){
                printf("CT:%2u BKX:%1u BKY:%1u TID:%2u YTID:%2u XTID:%2u GDX:%1u BDX:%1u GDY %1u BDY %1u\n",
                    cpu_calc_thread[y][x], cpu_block_x[y][x], cpu_block_y[y][x], cpu_thread[y][x], cpu_ythread[y][x],
                    cpu_xthread[y][x], cpu_grid_dimx[y][x], cpu_block_dimx[y][x],
                    cpu_grid_dimy[y][x], cpu_block_dimy[y][x]);
                ch = getchar();
            }
        }
        printf("Press any key to continue\n");
        ch = getchar();
    }

    hipFree(gpu_block_x);
    hipFree(gpu_block_y);
    hipFree(gpu_thread);
    hipFree(gpu_calc_thread);
    hipFree(gpu_xthread);
    hipFree(gpu_ythread);
    hipFree(gpu_grid_dimx);
    hipFree(gpu_block_dimx);
    hipFree(gpu_grid_dimy);
    hipFree(gpu_block_dimy);
}